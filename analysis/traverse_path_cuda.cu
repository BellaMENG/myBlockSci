#include "hip/hip_runtime.h"
#include "util.hpp"
#include <string>
#include <queue>
#include <fstream>
#include <chrono>

__constant__ Blockchain chain;
__constant__ unordered_set<Address> dest_addrs;

void printAddressFromString(Blockchain &h_chain, string address) {
    auto randomAddress = getAddressFromString(address, h_chain.getAccess());
    if (randomAddress) {
        cout << "scriptNum: " << randomAddress->scriptNum << endl;
        cout << "type: " << randomAddress->type << endl;
        cout << "toString: " << randomAddress->toString() << endl;
    }
}

void printOutputs(Blockchain &h_chain, string address) {
    auto randomAddress = getAddressFromString(address, h_chain.getAccess());
    if (randomAddress) {
        RANGES_FOR(auto input, (*randomAddress).getInputs()) {
            Transaction tx = input.transaction();
//            if (outAddr != *randomAddress)
            RANGES_FOR(auto out, tx.outputs()) {
                Address out_addr = out.getAddress();
                cout << out_addr.toString() << endl;
                
            }
        }
    }
}

bool findPath(Blockchain &h_chain, string src, string dest) {
    auto srcAddress = getAddressFromString(src, h_chain.getAccess());
    auto destAddress = getAddressFromString(dest, h_chain.getAccess());
    if ((!srcAddress) || (!destAddress)) {
        return false;
    }
    queue<Address> address_queue;
    address_queue.push(*srcAddress);
    while (!address_queue.empty()) {
        Address curr = address_queue.front();
        address_queue.pop();
        RANGES_FOR(auto input, curr.getInputs()) {
            Transaction tx = input.transaction();
            RANGES_FOR(auto out, tx.outputs()) {
                Address out_addr = out.getAddress();
                if (out_addr == *destAddress) {
                    cout << "found" << endl;
                    return true;
                }
                else
                    address_queue.push(out_addr);
            }
        }
    }
    cout << "doesn't exist" << endl;
    return false;
}

bool findPathGroups(Blockchain &h_chain, string src, unordered_set<Address> dests) {
    auto srcAddress = getAddressFromString(src, h_chain.getAccess());
    if (!srcAddress)
        return false;
    queue<Address> address_queue;
    unordered_set<Address> visited;
    
    address_queue.push(*srcAddress);
    while (!address_queue.empty()) {
        Address curr = address_queue.front();
        address_queue.pop();
        RANGES_FOR(auto input, curr.getInputs()) {
            Transaction tx = input.transaction();
            RANGES_FOR(auto out, tx.outputs()) {
                Address out_addr = out.getAddress();
                if (visited.find(out_addr) == visited.end()) {
                    if ((dests.find(out_addr) != dests.end())) {
                        cout << "found" << endl;
                        return true;
                    }
                    else {
                        address_queue.push(out_addr);
                        visited.insert(out_addr);
                    }
                }
            }
        }
    }
    cout << "doesn't exist" << endl;
    return false;
}

void read_addresses(Blockchain& h_chain, string file_path, int& num_addrs, unordered_set<Address>& addresses) {
    std::ifstream inputf(file_path, std::ifstream::in);
    char sharp;
    string addr;
    inputf >> sharp >> num_addrs;
    for (int i = 0; i < num_addrs; ++i) {
        inputf >> addr;
        auto dest_addr = getAddressFromString(addr, h_chain.getAccess());
        if (dest_addr)
            addresses.insert(*dest_addr);
    }
}

bool findPathRaw(Address& src, unordered_set<Address>& dests) {
    queue<Address> address_queue;
    unordered_set<Address> visited;
    
    address_queue.push(src);
    while (!address_queue.empty()) {
        Address curr = address_queue.front();
        address_queue.pop();
        RANGES_FOR(auto input, curr.getInputs()) {
            Transaction tx = input.transaction();
            RANGES_FOR(auto out, tx.outputs()) {
                Address out_addr = out.getAddress();
                if (visited.find(out_addr) == visited.end()) {
                    if ((dests.find(out_addr) != dests.end())) {
                        return true;
                    }
                    else {
                        address_queue.push(out_addr);
                        visited.insert(out_addr);
                    }
                }
            }
        }
    }
    return false;

}

uint32_t testFindPath(Blockchain &h_chain, uint32_t start, uint32_t pubkeyCount, unordered_set<Address>& dests) {
    queue<Address> address_queue;
    unordered_set<Address> visited;
    
    uint32_t trueResults = 0;
    uint32_t falseResults = 0;
    for (uint32_t i = start; i < start + pubkeyCount; ++i) {
        Address createAddr(i, AddressType::PUBKEYHASH, h_chain.getAccess());
        if (findPathRaw(createAddr, dests))
            trueResults++;
    }
    return trueResults;
}

__global__
void cuda_findPath(uint32_t start, uint32_t pubkeyCount) {
    
}

int main(int argc, const char* argv[]) {
    
    string chain_fp = argv[1];
    string src_addr = argv[2];
    string dest_addr = argv[3];
    string dest_addr_file_path = argv[4];
    int start = stoi(argv[5]);
    int num_addresses = stoi(argv[6]);
    Blockchain h_chain(chain_fp);
    
    hipMemcpyToSymbol(HIP_SYMBOL(*chain), *h_chain, sizeof(Blockchain));
    
//    findPath(chain, src_addr, dest_addr);
    
    int num_addrs;
    unordered_set<Address> h_dest_addrs;
    
    read_addresses(h_chain, dest_addr_file_path, num_addrs, h_dest_addrs);
    hipMemcpyToSymbol(HIP_SYMBOL(&dest_addr), h_dest_addrs, num_addrs*sizeof(Address));
    
    uint32_t pubkeyCount = h_chain.addressCount(AddressType::PUBKEYHASH);
    cout << "pubkey count: " << pubkeyCount << endl;
    if (num_addresses > pubkeyCount)
        num_addresses = pubkeyCount;
    
    hipDeviceReset();
    hipEvent_t cuda_start, cuda_end;
    float kernel_time;
    
    auto start_clock = chrono::high_resolution_clock::now();
    
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_end);
    hipEventRecord(cuda_start);

//    cuda_findPath<<<512, 8>>>();
    
    hipEventRecord(cuda_end);

    hipEventSynchronize(cuda_start);
    hipEventSynchronize(cuda_end);

    hipEventElapsedTime(&kernel_time, cuda_start, cuda_end);
    GPUErrChk(hipDeviceSynchronize());
    
    auto end_clock = chrono::high_resolution_clock::now();
    chrono::duration<double> diff = end_clock - start_clock;
    printf("Elapsed Time: %.9lf s\n", diff.count());
    printf("Number of positive results is %d\n", (int)trueResults);
    float avg = diff.count()/num_addresses;
    printf("Average query time is %.9lf s\n", avg);
    
    return 0;
}
