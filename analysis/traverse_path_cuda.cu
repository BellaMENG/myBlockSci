#include "hip/hip_runtime.h"
#include "util.hpp"
#include "traverse_path.h"
#include <string>
#include <queue>
#include <fstream>
#include <chrono>

__constant__ Blockchain chain;
__constant__ unordered_set<Address> dest_addrs;

__global__
void cuda_findPath(uint32_t start, uint32_t pubkeyCount) {
    
}

int main(int argc, const char* argv[]) {
    
    string chain_fp = argv[1];
    string src_addr = argv[2];
    string dest_addr = argv[3];
    string dest_addr_file_path = argv[4];
    int start = stoi(argv[5]);
    int num_addresses = stoi(argv[6]);
    Blockchain h_chain(chain_fp);
    
    hipDeviceReset();
    hipEvent_t cuda_start, cuda_end;
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_end);
    hipEventRecord(cuda_start);

    hipMemcpyToSymbol(HIP_SYMBOL(*chain), *h_chain, sizeof(Blockchain));
    
//    findPath(chain, src_addr, dest_addr);

    int num_addrs;
    unordered_set<Address> h_dest_addrs;
    
    read_addresses(h_chain, dest_addr_file_path, num_addrs, h_dest_addrs);
    hipMemcpyToSymbol(HIP_SYMBOL(&dest_addr), h_dest_addrs, num_addrs*sizeof(Address));
    
    uint32_t pubkeyCount = h_chain.addressCount(AddressType::PUBKEYHASH);
    cout << "pubkey count: " << pubkeyCount << endl;
    if (num_addresses > pubkeyCount)
        num_addresses = pubkeyCount;
    
    
    float kernel_time;
    
    auto start_clock = chrono::high_resolution_clock::now();

//    cuda_findPath<<<512, 8>>>();
    
    hipEventRecord(cuda_end);

    hipEventSynchronize(cuda_start);
    hipEventSynchronize(cuda_end);

    hipEventElapsedTime(&kernel_time, cuda_start, cuda_end);
    GPUErrChk(hipDeviceSynchronize());
    
    auto end_clock = chrono::high_resolution_clock::now();
    chrono::duration<double> diff = end_clock - start_clock;
    printf("Elapsed Time: %.9lf s\n", diff.count());
    printf("Number of positive results is %d\n", (int)trueResults);
    float avg = diff.count()/num_addresses;
    printf("Average query time is %.9lf s\n", avg);
    
    return 0;
}
